#include "hip/hip_runtime.h"
#include "cudamag.h"
#include <iostream>


__global__ void calcB(float* Bout, float* d_pts, int numPts)
{
    // threadIdx.x is the index of the point we're considering
    // blockIdx.x is the dimension (0 for x, 1 for y, 2 for z)
    for (int ii = 0; ii < numPts; ii++)
    {
        float distCubed = pow(pow(d_pts[3*(ii+threadIdx.x)]-d_pts[3*ii],2) + pow(d_pts[3*(ii+threadIdx.x)+1]-d_pts[3*ii+1],2) + pow(d_pts[3*(ii+threadIdx.x)+2]-d_pts[3*ii+2],2), -1.5);
        Bout[ii*numPts+threadIdx.x+numPts*numPts*blockIdx.x] = (d_pts[3*(ii+threadIdx.x)+blockIdx.x]-d_pts[3*ii+blockIdx.x]) * distCubed;
    }
}


CudaMag::CudaMag()
{
    numPts = 0;
    //numMagnets = 0;

    std::cout << "Magnet system created.\n";
}

CudaMag::~CudaMag()
{
    //hipFree(d_pts);
    //hipFree(d_areas);
    //hipFree(d_B);
}
/*
void CudaMag::addMagnet(Magnet* magnet)
{
    magnets.push_back(magnet);
}

void CudaMag::init()
{
    // Calculate total points
    for (int ii = 0; ii < magnets.size(); ii++) numPts += magnets[ii]->getNumPts();

    // Allocate memory
    hipMalloc(&d_pts, numPts*3*sizeof(float));
    hipMalloc(&d_areas, numPts*magnets.size()*sizeof(float));
    hipMemset(d_areas, 0, numPts*magnets.size()*sizeof(float));
    hipMalloc(&d_B, numPts*numPts*3*sizeof(float));
    hipMalloc(&d_sigma, numPts*sizeof(float));

    // Transfer data to GPU
    int ctr = 0;
    for (int ii = 0; ii < magnets.size(); ii++)
    {
        hipMemcpyAsync(d_pts+ctr, magnets[ii]->getPts(), magnets[ii]->getNumPts()*3*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_areas+ctr+numPts, magnets[ii]->getAreas(), magnets[ii]->getNumPts()*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_sigma+ctr, magnets[ii]->getSigma(), magnets[ii]->getNumPts()*sizeof(float), hipMemcpyHostToDevice);
        ctr += magnets[ii]->getNumPts();
    }
}


void CudaMag::calcBmat()
{
    calcB<<<3, numPts>>>(d_B, d_pts, numPts);
}

void CudaMag::solve()
{

}

*/